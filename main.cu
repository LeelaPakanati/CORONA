#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <ctime>
#include <nlohmann/json.hpp>

#include "./datatypes/location.cpp"
#include "./datatypes/person.cpp"
#include "./datatypes/disease.cpp"

// temporary
#define MOVEMENT_PROBABILITY .1

// variables
int SIMULATION_LENGTH = 365*24;
bool DEBUG;

// for convenience
using json = nlohmann::json;

void initialize(std::vector<Location*> *places, int numPeople, int numPlaces) {
	Location* loc_ptr;
	for(int i = 0; i < numPlaces; i++) {
		Location* loc_ptr = (Location*) malloc(sizeof(Location));
		Location loc;
		loc.interaction_level = 1.;
		*loc_ptr = loc;
		places->push_back(loc_ptr);
		//TODO: do something with duration function (inheritance?)
	}

	for(int i = 0; i < numPeople; i++) {
		Person* person_ptr = (Person*) malloc(sizeof(Person));
		Person person;
		person.infection_status = SUSCEPTIBLE;
		*person_ptr = person;
		Location* starting_loc = (*places)[rand() % places->size()];
		starting_loc->people_next_step.push_back(person_ptr);
	}

	for(int i = 0; i < numPlaces; i++) {
		std::clog << "Location " << i << " has " << (*places)[i]->people_next_step.size() << " people." << std::endl;
	}
}

void updateLocations(std::vector<Location*> *places, int n) {
	Location* loc_ptr;
	for (int loc_idx = 0; loc_idx < places->size(); loc_idx++) {
		loc_ptr = (*places)[loc_idx];
		loc_ptr->people.swap(loc_ptr->people_next_step);
		loc_ptr->people_next_step.clear();
	}
}

__global__ void spreadDisease(Location** dev_places, int num_places, int* place_num_people, placeDisease* dev_disease, unsigned long rand_seed) {
	Location* loc_ptr;
	Person* person_ptr;
	loc_ptr = &dev_places[blockIdx.x];
	int num_people = place_num_people[blockIdx.x];

	hiprandState_t state;
	cudarand_init(rand_seed, blockIdx.x*blockDim.x+threadIdx.x, 0, &state); 

	//determine spread of infection from infected to healthy
	__shared__ int has_sick = 0;
	
	for(int i = 0; i < num_people/blockDim.x+1; i++){
		int personIdx = i*blockDim.x + threadIdx.x;
		if(personIdx < num_people){
			person_ptr = loc_ptr->people[personIdx];
			// concurrency issue but only care if it 'ever' gets set to 1
			if ((person_ptr->infection_status == SICK) || (person_ptr->infection_status == CARRIER)) {
				has_sick = 1;
			}
		}
	}

	__syncthreads();

	// Propogate infections in places with infected people
	if(has_sick > 0) {
		for(int i = 0; i < num_people/blockDim.x+1; i++){
			int personIdx = i*blockDim.x + threadIdx.x;
			if(personIdx < num_people){
				person_ptr = loc_ptr->people[personIdx];
				if(person_ptr->infection_status == SUSCEPTIBLE){
					float infection_probability = disease->SPREAD_FACTOR * loc_ptr->interaction_level;
					float r = curand_unifrom(&state);
					if (r < infection_probability) {
						person_ptr->infection_status = CARRIER;
					}
				}
			}
		}
	}
}

void findNextLocations(std::vector<Location*> *places, int numPlaces) {
	Location* loc_ptr;
	Person* person_ptr;
	for (int loc_idx = 0; loc_idx < numPlaces; loc_idx++) {
		loc_ptr = (*places)[loc_idx];
		for (int person_idx = 0; person_idx < loc_ptr->people.size(); person_idx++) {
			person_ptr = loc_ptr->people[person_idx];
			float r = (float) rand() / RAND_MAX;
			if(r < MOVEMENT_PROBABILITY) {
				int new_loc = rand() % numPlaces;
				(*places)[new_loc]->people_next_step.push_back( person_ptr );
			} else {
				loc_ptr->people_next_step.push_back( person_ptr );
			}
		}
	}
}

__global__ void advanceInfection(std::vector<Location> places, int num_places, int* place_num_people, Disease* disease, int* susceptible, int* infected, int* recovered, int* deceased, unsigned long rand_seed){
	Location* loc_ptr;
	Person* person_ptr;
	loc_ptr = &dev_places[blockIdx.x];
	int num_people = place_num_people[blockIdx.x];

	hiprandState_t state;
	cudarand_init(rand_seed, blockIdx.x*blockDim.x+threadIdx.x, 0, &state); 

	for(int i = 0; i < num_people/blockDim.x+1; i++){
		int personIdx = i*blockDim.x + threadIdx.x;
		if(personIdx < num_people){
			person_ptr = loc_ptr->people[personIdx];
			switch (person_ptr->infection_status) {
				case CARRIER:
					// TODO: Normal Distribution around average times
					if (person_ptr->state_count > (int) disease->AVERAGE_INCUBATION_DURATION) {
						person_ptr->infection_status = SICK;
						person_ptr->state_count = 0;

						// TODO: death rate based on age
						float r = hiprand_normal(&state);
						person_ptr->to_die = (r < disease->DEATH_RATE);
					} else {
						person_ptr->state_count++;
					}
					break;

				case SICK:
					if (person_ptr->to_die) {
						if (person_ptr->state_count > disease->AVERAGE_TIME_DEATH)
							person_ptr->infection_status = DECEASED;
					} else {
						if (person_ptr->state_count > disease->AVERAGE_TIME_RECOVERY)
							person_ptr->infection_status = RECOVERED;
					}
					person_ptr->state_count++;
					break;
				default:
					break;
			}
		}
	}
}


void collectStatistics(std::vector<Location*> *places, int n, Disease* disease, int* susceptible, int* infected, int* recovered, int* deceased) {
	(*susceptible) = 0;
	(*infected) = 0;
	(*recovered) = 0;
	(*deceased) = 0;
	Location* loc_ptr;
	Person* person_ptr;
	for (int loc_idx = 0; loc_idx < places->size(); loc_idx++) {
		loc_ptr = (*places)[loc_idx];
		// Get number of sick people and set of susceptible people
		for (int person_idx = 0; person_idx < loc_ptr->people.size(); person_idx++) {
			person_ptr = loc_ptr->people[person_idx];
			switch (person_ptr->infection_status) {
				case SUSCEPTIBLE:
					(*susceptible)++;
					break;
				case CARRIER:
					(*infected)++;
					break;
				case SICK:
					(*infected)++;
					break;
				case RECOVERED:
					(*recovered)++;
					break;
				case DECEASED:
					(*deceased)++;
					break;
				default:
					break;
			}
		}
	}
}

int main(int argc, char** argv){

	// Parse arguments
	if (argc < 2){
		std::cerr << "Usage : " << argv[0] << " <input file>" << std::endl;
		return 0;
	}

	// Parse argument file
	std::string input_file_name = argv[1];
	std::clog << "Reading file " << input_file_name << " for starting conditions" << std::endl;

	// TODO: Add  more complex person/location config
	std::ifstream input_file(input_file_name);
	json input_json = json::parse(input_file);
	
	int pop_size = input_json.value("population_size", 0);
	int num_locs = input_json.value("num_locations", 0);
	DEBUG = input_json.value("debug", 0);

	srand(time(NULL));
	
	// All other references to these objects should be pointers or arrays of pointers
	std::vector<Location*> places;
	std::vector<Person*> people;

	initialize(&places, pop_size, num_locs);
	
	// Configure disease based on input argument
	json disease_json = input_json.value("disease", input_json);
	Disease disease(disease_json);

	int num_infected = input_json.value("initial_infected", 0);
	int person_to_infect;
	int location_to_infect;

	// Infect initial population.
	// note: people in less populated locations are more likely to be infected.
	// This should only matter for initially infecting large (>25%?) amounts of the population.
	for(int i = 0; i < num_infected; i++) {
		do {
			location_to_infect = rand() % num_locs;
			person_to_infect = rand() % places[location_to_infect]->people_next_step.size();
		} while(places[location_to_infect]->people_next_step[person_to_infect]->infection_status != SUSCEPTIBLE);
		places[location_to_infect]->people_next_step[person_to_infect]->infection_status = CARRIER;
		if(DEBUG) {
			std::clog << location_to_infect << " has an infected person" << std::endl;
		}
	}

	// Susciptible/Infected/Recovered/Deceased
	int num_susceptible, num_recovered, num_deceased;

	Location *loc_ptr;
	Person *person_ptr;

	dim3 dimGrid(num_locs, 1, 1);
	dim3 dimGrid(256, 1, 1);

	if(DEBUG) std::cout << "Susceptible,Infected,Recovered,Deceased" << std::endl;
	for(int hour = 0; num_infected > 0 && hour < SIMULATION_LENGTH; hour++) {
		updateLocations(&places, places.size());
		num_infected = num_susceptible = num_recovered = num_deceased = 0;
		collectStatistics(&places, places.size(), &disease, &num_susceptible, &num_infected, &num_recovered, &num_deceased);
		advanceInfection<<<dimGrid, dimBlock>>>(....);		
		spreadDisease<<<dimGrid, dimBlock>>>(....);		
		findNextLocations(&places, places.size());
		if(DEBUG) std::cout << num_susceptible << "," << num_infected << "," << num_recovered << "," << num_deceased << std::endl;
	}
}
