#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <ctime>
#include <cudarand.h>
#include <cudarand_kernel.h>
#include <nlohmann/json.hpp>

#include "./datatypes/location.cpp"
#include "./datatypes/person.cpp"
#include "./datatypes/disease.cpp"

// temporary
#define MOVEMENT_PROBABILITY .1

// variables
int SIMULATION_LENGTH = 365*24;
bool DEBUG;

// for convenience
using json = nlohmann::json;

void initialize(std::vector<Person> people, std::vector<Location> places, int numPeople, int numPlaces) {
	Location *loc_ptr;
	for(int i = 0; i < numPlaces; i++) {
		Location loc;
		loc_ptr = &loc;
		loc_ptr->interaction_level = 1.;
		places.push_back(*loc_ptr);
		//TODO: do something with duration function (inheritance?)
	}

	Person *person_ptr;
	for(int i = 0; i < numPeople; i++) {
		Person person;
		person_ptr = &person;
		person_ptr->infection_status = SUSCEPTIBLE;
		people.push_back(*person_ptr);
		places[rand() % places.size()].people_next_step.push_back(person_ptr);
	}

	for(int i = 0; i < numPlaces; i++) {
		std::clog << "Location " << i << " has " << places[i].people_next_step.size() << " people." << std::endl;
	}
}

void updateLocations(std::vector<Location> places, int n) {
	Location* loc_ptr;
	for (int loc_idx = 0; loc_idx < places.size(); loc_idx++) {
		loc_ptr = &places[loc_idx];
		loc_ptr->people.swap(loc_ptr->people_next_step);
		loc_ptr->people_next_step.clear();
	}
}

__global__ void spreadDisease(Location** dev_places, int num_places, int* place_num_people, placeDisease* dev_disease, unsigned long rand_seed) {
	Location* loc_ptr;
	Person* person_ptr;
	loc_ptr = &dev_places[blockIdx.x];
	int num_people = place_num_people[blockIdx.x];

	hiprandState_t state;
	cudarand_init(rand_seed, blockIdx.x*blockDim.x+threadIdx.x, 0, &state); 

	//determine spread of infection from infected to healthy
	__shared__ int has_sick = 0;
	
	for(int i = 0; i < num_people/blockDim.x; i++){
		int personIdx = i*blockDim.x + threadIdx.x;
		if(personIdx < num_people){
			person_ptr = loc_ptr->people[personIdx];
			// concurrency issue but only care if it 'ever' gets set to 1
			if ((person_ptr->infection_status == SICK) || (person_ptr->infection_status == CARRIER)) {
				num_sick = 1;
			}
		}
	}

	__syncthreads();

	// Propogate infections in places with infected people
	if(has_sick > 0) {
		for(int i = 0; i < num_people/blockDim.x; i++){
			int personIdx = i*blockDim.x + threadIdx.x;
			if(personIdx < num_people){
				person_ptr = loc_ptr->people[personIdx];

				// TODO: scale infection probability properly
				float infection_probability = disease->SPREAD_FACTOR * loc_ptr->interaction_level;
				float r = curand_unifrom(&state);
				if (r < infection_probability) {
					person_ptr->infection_status = CARRIER;
				}
			}
		}
	}
}

void findNextLocations(std::vector<Location> places, std::vector<Person> people, int numPlaces, int numPeople) {
	Location* loc_ptr;
	Person* person_ptr;
	for(int person_idx = 0; person_idx < numPeople; person_idx++){
		person_ptr = &people[person_idx];
		float r = (float) rand() / RAND_MAX;
		if(r < MOVEMENT_PROBABILITY) {
			int new_loc = rand() % numPlaces;
			places[new_loc].people_next_step.push_back( person_ptr );
		} else {
			loc_ptr->people_next_step.push_back( person_ptr );
		}
	}
}

__global__ void advanceInfection(std::vector<Location> places, int num_places, int* place_num_people, Disease* disease, int* susceptible, int* infected, int* recovered, int* deceased, unsigned long rand_seed){
	Location* loc_ptr;
	Person* person_ptr;
	loc_ptr = &dev_places[blockIdx.x];
	int num_people = place_num_people[blockIdx.x];

	hiprandState_t state;
	cudarand_init(rand_seed, blockIdx.x*blockDim.x+threadIdx.x, 0, &state); 

	for(int i = 0; i < num_people/blockDim.x; i++){
		int personIdx = i*blockDim.x + threadIdx.x;
		if(personIdx < num_people){
			person_ptr = loc_ptr->people[personIdx];
			switch (person_ptr->infection_status) {
				case CARRIER:
					// TODO: Normal Distribution around average times
					if (person_ptr->state_count > (int) disease->AVERAGE_INCUBATION_DURATION) {
						person_ptr->infection_status = SICK;
						person_ptr->state_count = 0;

						// TODO: death rate based on age
						float r = hiprand_normal(&state);
						person_ptr->to_die = (r < disease->DEATH_RATE);
					} else {
						person_ptr->state_count++;
					}
					break;

				case SICK:
					if (person_ptr->to_die) {
						if (person_ptr->state_count > disease->AVERAGE_TIME_DEATH)
							person_ptr->infection_status = DECEASED;
					} else {
						if (person_ptr->state_count > disease->AVERAGE_TIME_RECOVERY)
							person_ptr->infection_status = RECOVERED;
					}
					person_ptr->state_count++;
					break;
				default:
					break;
			}
		}
	}
}


void collectStatistics(std::vector<Location> places, int n, Disease* disease, int* susceptible, int* infected, int* recovered, int* deceased, unsigned long rand_seed){
	Location* loc_ptr;
	Person* person_ptr;
	loc_ptr = &dev_places[blockIdx.x];
	person_ptr = loc_ptr->people[threadIdx.x];

	switch (person_ptr->infection_status) {
		case SUSCEPTIBLE:
			(*susceptible)++;
			person_ptr->state_count = 0;
			break;
		case CARRIER:
			(*infected)++;

			// TODO: Normal Distribution around average times
			if (person_ptr->state_count > (int) disease->AVERAGE_INCUBATION_DURATION) {
				person_ptr->infection_status = SICK;
				person_ptr->state_count = 0;

				// TODO: death rate based on age
				float r = (float) rand() / RAND_MAX;
				if (r < disease->DEATH_RATE)
					person_ptr->to_die = true;
				else
					person_ptr->to_die = false;
			} else {
				person_ptr->state_count++;
			}
			break;
		case SICK:
			(*infected)++;

			if (person_ptr->to_die) {
				if (person_ptr->state_count > disease->AVERAGE_TIME_DEATH)
					person_ptr->infection_status = DECEASED;
			} else {
				if (person_ptr->state_count > disease->AVERAGE_TIME_RECOVERY)
					person_ptr->infection_status = RECOVERED;
			}
			person_ptr->state_count++;
			break;
		case RECOVERED:
			(*recovered)++;
			break;
		case DECEASED:
			(*deceased)++;
			break;
		default:
			break;
	}
		
	
}

int main(int argc, char** argv){

	// Parse arguments
	if (argc < 2){
		std::cerr << "Usage : " << argv[0] << " <input file>" << std::endl;
		return 0;
	}

	// Parse argument file
	std::string input_file_name = argv[1];
	std::clog << "Reading file " << input_file_name << " for starting conditions" << std::endl;

	// TODO: Add  more complex person/location config
	std::ifstream input_file(input_file_name);
	json input_json = json::parse(input_file);
	
	int pop_size = input_json.value("population_size", 0);
	int num_locs = input_json.value("num_locations", 0);
	DEBUG = input_json.value("debug", 0);

	srand(time(NULL));
	
	// All other references to these objects should be pointers or arrays of pointers
	std::vector<Location> places;
	std::vector<Person> people;

	initialize(people, places, pop_size, num_locs);
	
	// Configure disease based on input argument
	json disease_json = input_json.value("disease", input_json);
	Disease disease(disease_json);

	int num_infected = input_json.value("initial_infected", 0);
	int person_to_infect;
	int location_to_infect;

	// Infect initial population.
	// note: people in less populated locations are more likely to be infected.
	// This should only matter for initially infecting large (>25%?) amounts of the population.
	for(int i = 0; i < num_infected; i++) {
		do {
			location_to_infect = rand() % num_locs;
			person_to_infect = rand() % places[location_to_infect].people_next_step.size();
		} while(people[person_to_infect].infection_status != SUSCEPTIBLE);
		people[person_to_infect].infection_status = CARRIER;
		if(DEBUG) {
			std::clog << location_to_infect << " has an infected person" << std::endl;
		}
	}

	// Susciptible/Infected/Recovered/Deceased
	int num_susceptible = pop_size - num_infected;
	int num_recovered = 0;
	int num_deceased = 0;

	Location *loc_ptr;
	Person *person_ptr;

	if(DEBUG) std::cout << "Susceptible,Infected,Recovered,Deceased" << std::endl;
	for(int hour = 0; num_infected > 0 && hour < SIMULATION_LENGTH; hour++) {
		updateLocations(places, places.size());
		spreadDisease(places, places.size(), &disease);
		findNextLocations(places, people, places.size(), people.size());
		num_infected = num_susceptible = num_recovered = num_deceased = 0;
		collectStatistics(places, places.size(), &disease, &num_susceptible, &num_infected, &num_recovered, &num_deceased);
		if(DEBUG) std::cout << num_susceptible << "," << num_infected << "," << num_recovered << "," << num_deceased << std::endl;
	}
}
